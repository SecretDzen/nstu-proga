#include <hip/hip_runtime.h>

#include <iostream>

int main() {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop, 0);
  printf("Device name: %s\n", dev_prop.name);

  printf("Max threads dimensions: x = %d, y = %d, z = %d\n",
         dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1],
         dev_prop.maxThreadsDim[2]);

  printf("Max grid size: x = %d, y = %d, z = %d\n", dev_prop.maxGridSize[0],
         dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
}