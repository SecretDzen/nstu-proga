#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
using namespace std;

#define MIN_SIZE 50
#define MAX_SIZE 2000
#define STEP 50

int N = (MAX_SIZE - MIN_SIZE) / STEP + 1;

double* create_matrix(int size) {
  double* matrix = (double*)malloc(sizeof(double) * size * size);

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      matrix[i * size + j] = rand() % 100 + 1;
    }
  }

  return matrix;
}

double get_det_gpu(double* matrix, int size) {
    dim3 BLOCKS_1D(16);
  dim3 THREADS_1D(32);
  dim3 BLOCKS_2D(16, 16);
  dim3 THREADS_2D(32, 32);

  double det = 1.0;
  double dev_det = 1.0;
  double* dev_matrix;
  int swap_count = 0;

  hipMalloc(&dev_matrix, sizeof(double) * size * size);
  hipMemcpy(dev_matrix, matrix, sizeof(double) * size * size,
             hipMemcpyHostToDevice);

  for (int i = 0; i < size; i++) {
    max_in_column<<<1, 1>>>(dev_matrix, size, i);
    swap_lines<<<BLOCKS_1D, THREADS_1D>>>(dev_matrix, size, i);
    subtract_below<<<BLOCKS_2D, THREADS_2D>>>(dev_matrix, dev_identity, n, i);
    nullify_below<<<BLOCKS_1D, THREADS_1D>>>(dev_matrix, n, i);
    hipDeviceSynchronize();

  }

  hipMemcpy(&det, &dev_det, sizeof(long double), hipMemcpyDeviceToHost);

  hipFree(dev_matrix);
  return det * pow(-1, swap_count);
}

int main() {
  srand(time(NULL));

  for (int size = MIN_SIZE, i = 0; size <= MAX_SIZE; size += STEP, i++) {
    double* matrix = create_matrix(size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    get_det_gpu(matrix, size);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(matrix);
    printf("%d ", size);
    printf("%.3f\n", time / CLOCKS_PER_SEC);
  }
  return 0;
}
