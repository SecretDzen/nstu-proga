// Libraries
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
using namespace std;

int MAX_THREADS_PER_BLOCK;

// Constants
#define startSize 100
#define maxSize 400
#define step 10

int N = (maxSize - startSize) / step + 1;  // quantity of elements
// Filling up matrix with elements in range [1, 100]
float* fillUpMatrix(int size) {
  float* matrix = (float*)malloc(size * size * sizeof(float));

  for (int i = 0; i < size; ++i) {
    for (int j = 0; j < size; ++j) {
      matrix[i * size + j] = rand() % 100 + 1;
    }
  }

  return matrix;
}

__global__ void multiplyDetWithElement(long double* det, float* matrix,
                                       int currentDiagonalElemIndex) {
  *det *= matrix[currentDiagonalElemIndex];  // multiplying determinant with
                                             // diagonal element
}

__global__ void fillCoefsArray(float* coefs, float* matrix, int size,
                               int currentDiagonalElemIndex,
                               int startNumber = 0) {
  int i = startNumber + blockDim.x * blockIdx.x +
          threadIdx.x;  // unique index for each coefficient
  int elemToZeroIndex = currentDiagonalElemIndex +
                        size * (i + 1);  // element that we want to cast to null

  coefs[(elemToZeroIndex / size) - 1] =
      -matrix[elemToZeroIndex] / matrix[currentDiagonalElemIndex];
}

__global__ void multiplyElemWithCoef(float* matrix, int size,
                                     int currentDiagonalElemRow, float* coefs,
                                     int startNumber = 0) {
  int number = startNumber + blockDim.x * blockIdx.x + threadIdx.x;
  int columnsCount = size - currentDiagonalElemRow;

  int row = currentDiagonalElemRow + 1 + (number / columnsCount);
  int column = currentDiagonalElemRow + (number % columnsCount);

  matrix[row * size + column] +=
      coefs[row - 1] * matrix[currentDiagonalElemRow * size + column];
}

void getNumberOfBlocksAndThreads(int elemsCount, int* blocks, int* threads,
                                 int* remains) {
  if (elemsCount < MAX_THREADS_PER_BLOCK) {
    *blocks = 1;
    *threads = elemsCount;
  } else {
    *blocks = elemsCount / MAX_THREADS_PER_BLOCK;
    *threads = MAX_THREADS_PER_BLOCK;
  }
  *remains = elemsCount - *blocks * *threads;
}

hipError_t allocateMemory(float* matrix, float** gpuMatrix, int size,
                           float** gpuCoefs, long double** gpuDet) {
  // Allocating memory on GPU for determinant, matrix(1-dimension), coefficients
  hipError_t status = hipMalloc(gpuMatrix, size * size * sizeof(float));
  status = hipMemcpy(*gpuMatrix, matrix, size * size * sizeof(float),
                      hipMemcpyHostToDevice);

  status = hipMalloc(gpuCoefs, (size - 1) * sizeof(float));

  status = hipMalloc(gpuDet, 1 * sizeof(long double));
  long double det = 1;
  status = hipMemcpy(*gpuDet, &det, 1 * sizeof(long double),
                      hipMemcpyHostToDevice);

  return status;
}

long double gaussMethod(float* matrix, int size) {
  long double det = 1;

  long double* _det = nullptr;
  float* _matrix = nullptr;
  float* _coefs = nullptr;
  int error = 0;

  if (allocateMemory(matrix, &_matrix, size, &_coefs, &_det) != hipSuccess) {
    error = 1;
  }

  for (int i = 0; i < size && !error; i++) {
    int curDiagonalElemIndex =
        i * size + i;  // Index of current diagonal element
    multiplyDetWithElement<<<1, 1>>>(
        _det, _matrix,
        curDiagonalElemIndex);  // Multiplying determinant with diagonal element

    int blocksCount, threadsCount, remains;
    getNumberOfBlocksAndThreads(size - i - 1, &blocksCount, &threadsCount,
                                &remains);

    fillCoefsArray<<<blocksCount, threadsCount>>>(_coefs, _matrix, size,
                                                  curDiagonalElemIndex);
    fillCoefsArray<<<1, remains>>>(_coefs, _matrix, size, curDiagonalElemIndex,
                                   blocksCount * threadsCount);

    int elemsCount = (size - 1 - i) *
                     (size - i);  // Elems that will be affected by iteration
    getNumberOfBlocksAndThreads(elemsCount, &blocksCount, &threadsCount,
                                &remains);

    hipDeviceSynchronize();  // waiting for GPU done calculations

    multiplyElemWithCoef<<<blocksCount, threadsCount>>>(_matrix, size, i,
                                                        _coefs);
    multiplyElemWithCoef<<<1, remains>>>(_matrix, size, i, _coefs,
                                         blocksCount * threadsCount);

    hipDeviceSynchronize();  // waiting for GPU done calculations
  }

  hipMemcpy(&det, &_det[0], sizeof(long double), hipMemcpyDeviceToHost);

  hipFree(_matrix);
  hipFree(_det);
  hipFree(_coefs);
  return det;
}

int main() {
  srand(time(NULL));
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);  // getting blocks size from GPU
  MAX_THREADS_PER_BLOCK = deviceProp.maxThreadsDim[0];

  for (int size = startSize, i = 0; size <= maxSize; size += step, i++) {
    float* matrix = fillUpMatrix(size);  // filling up the matrix

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);  // CUDA timers for calculating time

    //------
    hipEventRecord(start, 0);
    gaussMethod(matrix, size);  // processing Gauss-method
    hipEventRecord(stop, 0);
    //------

    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(matrix);
    printf("%d ", size);                      // matrix size
    printf("%.3f\n", time / CLOCKS_PER_SEC);  // calculation time
  }
  return 0;
}
